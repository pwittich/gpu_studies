#include "hip/hip_runtime.h"
// -*-c++-*-
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <algorithm>
#include <numeric>
#include <list>
#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>

#include "Matriplex.h"

using namespace Matriplex;

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, 
		      bool abort=true) 
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, 
	      line);
      if (abort) exit(code);
   }
}

// this approach runs into the 48 kB limit per shared memory
// for 6x6 matrices this means something like 1300 matrices, so
// for a matrix muptiplication something like 450 size of matriplex.
template<typename T, idx_t DIM1, idx_t DIM2, idx_t DIM3, idx_t N>
__global__ void matrixkern(const T *d1, 
			   const T *d2,
			   T *d3) 
{
  const int gti = blockIdx.x * blockDim.x + threadIdx.x;
  const int gStride = blockDim.x * gridDim.x;

  // copy data into matriplex
  Matriplex::MPlex<float, DIM1, DIM2, N> d_matrices1;
  Matriplex::MPlex<float, DIM2, DIM3, N> d_matrices2;


  // convert random data to matriplex
  for ( idx_t i = gti; i < N; i += gStride ) {
    //printf("gti = %d, i = %d, dest = %p\n", gti, i, d1);
    d_matrices1.CopyIn(i, d1+i*d_matrices1.kSize);
    d_matrices2.CopyIn(i, d2+i*d_matrices2.kSize);
  }

  Matriplex::Matriplex<float, DIM1, DIM3, N> d_result;
  // do matrix multiplication
  MultiplyGeneralStride(d_matrices1, d_matrices2, d_result, gti, gStride);

  // copy result back
  for ( idx_t i = gti; i < N; i += gStride ) 
    d_result.CopyOutPlex(i, d3);
  //d_result.CopyOut(i, d3+i*d_result.kSize);

}



int main()
{

  int num_devices, device;
  CUDA_SAFE_CALL(hipGetDeviceCount(&num_devices));
  printf("This many devices: %d\n", num_devices);
  int max_multiprocessors = -1, max_device = -1;
  hipDeviceProp_t best_prop;
  for ( device = 0; device < num_devices; ++device ) {
    hipDeviceProp_t properties;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&properties, device));
    if ( max_multiprocessors < properties.multiProcessorCount ) {
      max_multiprocessors = properties.multiProcessorCount;
      max_device = device;
      best_prop = properties;
    }
  }
  if ( max_device >=0 )
    hipSetDevice(max_device);
  else  {
    printf("problem finding a good device! aborting.\n");
    return 1;
  }
  printf("# Running on device %d (name %s)\n", max_device, best_prop.name);

  // set the memory limits on the device

  const int NBLOCKS = 1;
  const int DIM1 = 4;
  const int DIM2 = 4;
  const int DIM3 = 4;
  //const int N = 103-6;
  const int N = 256;
  const int nmatrix1 = DIM1*DIM2*N;
  const int nmatrix2 = DIM2*DIM3*N;
  const int nmatrixres = DIM1*DIM3*N;
  printf("Size of memory required: %5.1f kB\n",
	 NBLOCKS * sizeof(float)*(nmatrix1+nmatrix2+nmatrixres)/1024.);

  // get the heap size
  size_t curSize = 0;
  hipDeviceGetLimit(&curSize, hipLimitMallocHeapSize);
  curSize *=5.0;
  hipError_t err = hipDeviceSetLimit(hipLimitMallocHeapSize, curSize);
  if ( err != hipSuccess ) {
    printf("failed to set heap size to %d\n", curSize);
    return 1;
  }
  hipDeviceGetLimit(&curSize, hipLimitMallocHeapSize);
  printf("Current size: %5.0f kB\n", curSize/1024.);

  // fill matrices with random data
  float mres[nmatrixres];
  float mres_gpu[nmatrixres];
  memset(mres, 0,nmatrixres*sizeof(float));
  memset(mres_gpu, 0,nmatrixres*sizeof(float));



  hipDeviceSynchronize();

  // these vectors hold the pre-matriplex matrices
  std::vector<float> h_pos1(nmatrix1);
  std::vector<float> h_pos2(nmatrix2);
  std::fill(h_pos2.begin(), h_pos2.end(),0);

  // space on gpu for the inputs and outputs
  float *d_f1 = 0;
  float *d_f2 = 0;
  float *d_fres = 0;

  hipMalloc(&d_f1, nmatrix1*sizeof(float));
  hipMalloc(&d_f2, nmatrix2*sizeof(float));
  hipMalloc(&d_fres, nmatrixres*sizeof(float));

  float *h_f1 = &h_pos1[0]; 
  float *h_f2 = &h_pos2[0]; 
  

  srand(123213UL);



  for ( auto & i : h_pos1 ) {
    i = rand()*20./RAND_MAX;
  }
  for ( auto & i : h_pos2 ) {
    i = rand()*20./RAND_MAX;
  }





  // copy to GPU
  printf("copying to GPU .... \n");
  hipMemcpyAsync(d_f1, h_f1, sizeof(float)*nmatrix1, hipMemcpyHostToDevice);
  hipMemcpyAsync(d_f2, h_f2, sizeof(float)*nmatrix2, hipMemcpyHostToDevice);




  Matriplex::MPlex<float, DIM1, DIM2, N> h_matrices1;
  Matriplex::MPlex<float, DIM2, DIM3, N> h_matrices2;
  memset(h_matrices1.fArray, 0, h_matrices1.kTotSize*sizeof(float));
  memset(h_matrices2.fArray, 0, h_matrices2.kTotSize*sizeof(float));

  // // convert random data to matriplex
  for ( idx_t i = 0; i < N; ++i ) {
    h_matrices1.CopyIn(i, h_pos1.data()+i*h_matrices1.kSize);
    h_matrices2.CopyIn(i, h_pos2.data()+i*h_matrices2.kSize);
  }


 Matriplex::Matriplex<float, DIM1, DIM3, N> h_result;
 MultiplyGeneral(h_matrices1, h_matrices2, h_result);
 // convert resulting data from matriplex
  for ( idx_t i = 0; i < N; ++i ) 
    h_result.CopyOut(i, mres+i*(h_result.kSize));

  // result is now in d_fres
  for ( int i = 0; i < 10; ++i ) 
  matrixkern<float, DIM1, DIM2, DIM3,N><<<NBLOCKS,128>>>(d_f1,d_f2, 
						    d_fres );
  //hipDeviceSynchronize();
  hipDeviceSynchronize();
  // check for error. this catches a kernel launch error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess) {
     // print the CUDA error message and exit
     printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,hipGetErrorString(error));
     exit(-1);
  }
  // copy result back
  CUDA_SAFE_CALL(hipMemcpy(mres_gpu,d_fres,sizeof(float)*nmatrixres, hipMemcpyDeviceToHost));



  printf("i:cpu\tgpu\n");
  for (int i = 0;i<nmatrixres; ++i ) {
    printf("%d: (%d) %8.3f\t%8.3f %s\n", i, int(i/h_result.kSize),mres[i], mres_gpu[i], 
 	   (fabs(mres[i]-mres_gpu[i])<1.0e-3)?"":"<<<");
  }

   
  
  return 0;
}
