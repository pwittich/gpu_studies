#include "hip/hip_runtime.h"
// -*-c++-*-
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <algorithm>
#include <numeric>
#include <list>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#if defined(__HIPCC__) // NVCC
#define MY_ALIGN(n) __align__(n)
#elif defined(__GNUC__) // GCC
#define MY_ALIGN(n) __attribute__((aligned(n)))
#elif defined(_MSC_VER) // MSVC
#define MY_ALIGN(n) __declspec(align(n))
#else
  #error "Please provide a definition for MY_ALIGN macro for your host compiler!"
#endif

float4
__device__ __host__ operator*(const float4 a, const float4 b)
{
  return make_float4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w);
}
float4
__device__ __host__ operator-(const float4 a, const float4 b)
{
  return make_float4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w);
}

float4
__device__ __host__ operator*(const float a, const float4 b)
{
  return make_float4(a*b.x, a*b.y, a*b.z, a*b.w);
}

float4
__device__ __host__ operator+=(const float4 a, float4 b)
{
  b = make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
  return b;
}

float4
__device__ __host__ operator+(const float4 a, const float4 b)
{
  return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

using namespace thrust;


__device__ __host__ 
void update_particle(const size_t i, const size_t nparticle, 
		     const float4 * pos_old,
		     float4 * pos_new, 
		     float4 * vel)
{
  const float dt = 0.1;
  const float4 dt0 =make_float4(dt,dt,dt,0.0f);
  const float eps = 0.0001;
  float4 p = pos_old[i];
  float4 v = vel[i];
  float4 a = make_float4(0.0f,0.0f,0.0f,0.0f);
  // does this include self-interaction?
  for(size_t j = 0; j < nparticle; ++j  ) { // inner loop over particles
    if ( i == j ) continue;
    const float4 p2 = pos_old[j]; //Read a particle position 
    float4 d = p2 - p;
    float invr = 1./sqrt(d.x*d.x + d.y*d.y + d.z*d.z + eps);
    float f = p2.w*invr*invr*invr;
    a += f*d; // Accumulate acceleration 
  }
  
  p += dt0*v + 0.5f*dt0*dt0*a;
  v += dt0*a;
  
  pos_new[i] = p;
  vel[i] = v;
  
}

// SOA - sorta?
struct MY_ALIGN(16) wrapper_t  {
  int toggle; // pos1->pos2 or pos2->pos1
  size_t nparticle;
  float4 *pos1, *vel, *pos2;
};


class MY_ALIGN(16) functor_thrust {
private:
  const struct wrapper_t _wrapper;
public:
  __host__ __device__
   functor_thrust( struct wrapper_t w ):
     _wrapper(w)
  {}
  
  __device__ __host__ 
  void operator()(const int i) const 
  {
    if ( _wrapper.toggle ) 
      update_particle(i, _wrapper.nparticle, _wrapper.pos1, _wrapper.pos2, _wrapper.vel);
    else
      update_particle(i, _wrapper.nparticle, _wrapper.pos2, _wrapper.pos1, _wrapper.vel);
      
  }

};

#define CUDA_SAFE_CALL(call) call

int main()
{


  int num_devices, device;
  CUDA_SAFE_CALL(hipGetDeviceCount(&num_devices));
  printf("This many devices: %d\n", num_devices);
  int max_multiprocessors = -1, max_device = -1;
  hipDeviceProp_t best_prop;
  for ( device = 0; device < num_devices; ++device ) {
    hipDeviceProp_t properties;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&properties, device));
    if ( max_multiprocessors < properties.multiProcessorCount ) {
      max_multiprocessors = properties.multiProcessorCount;
      max_device = device;
      best_prop = properties;
    }
  }
  if ( max_device >=0 )
    hipSetDevice(max_device);
  else  {
    printf("problem finding a good device! aborting.\n");
    return 1;
  }
  printf("# Running on device %d (name %s)\n", max_device, best_prop.name);



  //uint64_t        t0, t1, t2;
  int nparticle = 8; /* MUST be a nice power of two for simplicity */
  const int nstep = 3;
  //int nburst = 20; /* MUST divide the value of nstep without remainder */
  //int nthread = 64; /* chosen for ATI Radeon HD 5870 */

  const float dt = 0.1;
  const float eps = 0.0001;


  thrust::device_vector<int> d_ivals(nparticle);
  thrust::host_vector<int> h_ivals(nparticle);
  thrust::sequence(d_ivals.begin(), d_ivals.end()); // 0..nparticle-1
  printf("copying from GPU\n");
  h_ivals = d_ivals;
  //return 0;

  thrust::host_vector<float4> h_pos1(nparticle), h_pos2(nparticle), h_vel(nparticle);
  thrust::device_vector<float4> d_pos1(nparticle), d_pos2(nparticle), d_vel(nparticle);

  
  const float4 dt0 = make_float4(dt,dt,dt,0.0f);

  printf("making particles .... \n");
  srand(1232773);
  for ( int i = 0; i < nparticle; ++i ){
    h_pos1[i] = make_float4(100.*rand()/RAND_MAX  - 50.,
			    100.*rand()/RAND_MAX  - 50.,
			    100.*rand()/RAND_MAX - 50.,
			    10.*rand()/RAND_MAX // mass
			    );
    h_pos2[i] = make_float4(0.f,0.f,0.f,0.f);
    h_vel[i]  = make_float4(0.f,0.f,0.f,0.f);
  }
  for ( int i = 0; i < nparticle; ++i ) {
    printf("%d\t%f\n", i,h_pos1[i]);
  }

  struct wrapper_t wrapper;
  struct wrapper_t h_wrapper;
  // copy to GPU
  printf("copying to GPU .... 1\n");
  d_pos1 = h_pos1;
  printf("copying to GPU .... 2\n");
  d_pos2 = h_pos2;
  printf("copying to GPU .... 3\n");
  d_vel  = h_vel;
  printf("done copying to GPU .... \n");
  wrapper.pos1 = thrust::raw_pointer_cast(d_pos1.data());
  wrapper.pos2 = thrust::raw_pointer_cast(d_pos2.data());
  wrapper.vel  = thrust::raw_pointer_cast(d_vel .data());
  wrapper.nparticle = nparticle;
  wrapper.toggle = 0;

  h_wrapper.pos1 = thrust::raw_pointer_cast(h_pos1.data());
  h_wrapper.pos2 = thrust::raw_pointer_cast(h_pos2.data());
  h_wrapper.vel  = thrust::raw_pointer_cast(h_vel .data());
  h_wrapper.nparticle = nparticle;
  h_wrapper.toggle = 0;



  int which = 8;

  
  // loop over time steps
  printf("Starting loop \n");
  for ( int istep = 0; istep<nstep; ++istep ) {
    printf("istep = %d,\n",istep);
    if ( istep%2==0 ) {
      wrapper.toggle = 1;
    }
    else {
      wrapper.toggle = 0;
    }
    try {
      thrust::for_each(d_ivals.begin(), d_ivals.end(), functor_thrust(wrapper));
    }
    catch(thrust::system_error &e) {
      printf("error: %s\n", e.what());
    }
    
    // if ( istep%2==0 ) {
    //   h_wrapper.toggle = 1;
    // }
    // else {
    //   h_wrapper.toggle = 0;
    // }
    // std::for_each(h_ivals.begin(), h_ivals.end(), functor_thrust(h_wrapper));
    
    // tbb::parallel_for(blocked_range<size_t>(0,nparticle),
    //  		      functor_tbb(pos_old, pos_new, &vel,nparticle));
  }


  //t2 = mach_absolute_time();
  printf("done.\n");
  // struct mach_timebase_info info;
  // mach_timebase_info(&info);
  // double          t = 1e-9 * (t2 - t1) * info.numer / info.denom;
  // printf("Time spent = %g\n", t);


  //float4_t endpos = pos1[which];

  //printf("End:   particle %d x=%f, y=%f, z=%f, m=%f\n",
  // which, pos1[which].x, pos1[which].y, pos1[which].z, pos1[which].w);

  // //float4_t sep = endpos-startpos;;
  // float distance = sqrt(sep.x*sep.x + sep.y*sep.y + sep.z*sep.z);
  // printf("Distance travelled = %g\n", distance);

  h_pos1 = d_pos1; // copy back

  //dump
  for ( size_t i = 0; i < nparticle; ++i ) {
    printf("Final: particle %d x=%f, y=%f, z=%f, m=%f\n",
	 i, h_pos1[i].x, h_pos1[i].y, h_pos1[i].z, h_pos1[i].w);
  }


  return 0;
}
